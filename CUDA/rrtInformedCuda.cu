#include "hip/hip_runtime.h"
// rrtInformedCuda.cu
#include "rrtInformedCuda.h"
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include "cudaRRTUtils.h"
#include "cudaRRTKernels.h"
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <fstream>
#include <iostream>
#include <chrono>
#include <algorithm>
#include <cmath>

// Destructor implementation
RRTInformedCudaData::~RRTInformedCudaData() {
    // Free device memory
    if (d_nodeX) hipFree(d_nodeX);
    if (d_nodeY) hipFree(d_nodeY);
    if (d_nodeParent) hipFree(d_nodeParent);
    if (d_nodeTime) hipFree(d_nodeTime);
    if (d_nodeCost) hipFree(d_nodeCost);
    
    if (d_obstacleX) hipFree(d_obstacleX);
    if (d_obstacleY) hipFree(d_obstacleY);
    if (d_obstacleWidth) hipFree(d_obstacleWidth);
    if (d_obstacleHeight) hipFree(d_obstacleHeight);
    
    if (d_randStates) hipFree(d_randStates);
}

// Kernel implementations are moved to cudaRRTKernels.h


// Function to initialize CUDA resources
void initCudaRRTInformed(RRTInformedCudaData& data, int maxNodes, int numObstacles, int numThreads) {
    // Allocate memory for nodes
    data.d_nodeCapacity = maxNodes;
    CUDA_CHECK(hipMalloc(&data.d_nodeX, maxNodes * sizeof(float)));
    CUDA_CHECK(hipMalloc(&data.d_nodeY, maxNodes * sizeof(float)));
    CUDA_CHECK(hipMalloc(&data.d_nodeParent, maxNodes * sizeof(int)));
    CUDA_CHECK(hipMalloc(&data.d_nodeTime, maxNodes * sizeof(float)));
    CUDA_CHECK(hipMalloc(&data.d_nodeCost, maxNodes * sizeof(float)));
    
    // Allocate memory for obstacles
    if (numObstacles > 0) {
        CUDA_CHECK(hipMalloc(&data.d_obstacleX, numObstacles * sizeof(float)));
        CUDA_CHECK(hipMalloc(&data.d_obstacleY, numObstacles * sizeof(float)));
        CUDA_CHECK(hipMalloc(&data.d_obstacleWidth, numObstacles * sizeof(float)));
        CUDA_CHECK(hipMalloc(&data.d_obstacleHeight, numObstacles * sizeof(float)));
    }
    
    // Initialize random states
    CUDA_CHECK(hipMalloc(&data.d_randStates, numThreads * sizeof(hiprandState)));
    
    int blocks = (numThreads + BLOCK_SIZE - 1) / BLOCK_SIZE;
    initRandStatesKernel<<<blocks, BLOCK_SIZE>>>(data.d_randStates, 
                                               static_cast<unsigned long>(time(nullptr)));
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Initialize best cost to infinity
    data.h_bestCost = FLT_MAX;
    data.h_goalNodeIndex = -1;
}

// Function to clean up CUDA resources
void cleanupCudaRRTInformed(RRTInformedCudaData& data) {
    // Most cleanup is handled by the destructor
    // This function is provided for explicit cleanup if needed
    hipDeviceSynchronize();
}

// Function to find nearest node using CUDA
int findNearestCuda(RRTInformedCudaData& data, float x, float y) {
    if (data.h_nodeCount == 0) {
        return -1;
    }
    
    // Allocate memory for distances
    float* d_distances;
    CUDA_CHECK(hipMalloc(&d_distances, data.h_nodeCount * sizeof(float)));
    
    // Calculate number of blocks needed
    int blocks = (data.h_nodeCount + BLOCK_SIZE - 1) / BLOCK_SIZE;
    
    // Launch kernel to compute distances
    findNearestKernel<<<blocks, BLOCK_SIZE>>>(data.d_nodeX, data.d_nodeY, data.h_nodeCount, 
                                          x, y, d_distances);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Find minimum distance index using Thrust
    thrust::device_ptr<float> thrust_distances(d_distances);
    thrust::device_ptr<float> min_distance_ptr = thrust::min_element(
        thrust_distances, thrust_distances + data.h_nodeCount);
    
    // Get the index of the minimum element
    int minIndex = static_cast<int>(min_distance_ptr - thrust_distances);
    
    // Free memory
    CUDA_CHECK(hipFree(d_distances));
    
    return minIndex;
}

// Function to find nodes within a radius using CUDA
std::vector<int> findNodesInRadiusCuda(RRTInformedCudaData& data, float x, float y, float radius) {
    std::vector<int> result;
    
    if (data.h_nodeCount == 0) {
        return result;
    }
    
    // Allocate memory for results
    int* d_inRadius;
    CUDA_CHECK(hipMalloc(&d_inRadius, data.h_nodeCount * sizeof(int)));
    CUDA_CHECK(hipMemset(d_inRadius, 0, data.h_nodeCount * sizeof(int)));
    
    // Calculate number of blocks needed
    int blocks = (data.h_nodeCount + BLOCK_SIZE - 1) / BLOCK_SIZE;
    
    // Launch kernel to find nodes in radius
    float radiusSq = radius * radius;
    findNodesInRadiusKernel<<<blocks, BLOCK_SIZE>>>(
        data.d_nodeX, data.d_nodeY, data.h_nodeCount,
        x, y, radiusSq, d_inRadius);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Copy results back to host
    std::vector<int> h_inRadius(data.h_nodeCount);
    CUDA_CHECK(hipMemcpy(h_inRadius.data(), d_inRadius, 
                        data.h_nodeCount * sizeof(int), hipMemcpyDeviceToHost));
    
    // Build result vector
    for (int i = 0; i < data.h_nodeCount; i++) {
        if (h_inRadius[i] == 1) {
            result.push_back(i);
        }
    }
    
    // Free memory
    CUDA_CHECK(hipFree(d_inRadius));
    
    return result;
}

// Function to choose best parent using CUDA
int chooseBestParentCuda(RRTInformedCudaData& data, float x, float y, 
                       const std::vector<int>& neighbors) {
    if (neighbors.empty()) {
        return -1;
    }
    
    int neighborCount = neighbors.size();
    
    // Allocate memory for device arrays
    int* d_neighbors;
    float* d_costToNew;
    float* d_minCost;
    int* d_bestParent;
    
    CUDA_CHECK(hipMalloc(&d_neighbors, neighborCount * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_costToNew, neighborCount * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_minCost, sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_bestParent, sizeof(int)));
    
    // Initialize best cost to a large value and best parent to -1
    float initialCost = FLT_MAX;
    int initialParent = -1;
    CUDA_CHECK(hipMemcpy(d_minCost, &initialCost, sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_bestParent, &initialParent, sizeof(int), hipMemcpyHostToDevice));
    
    // Copy neighbors to device
    CUDA_CHECK(hipMemcpy(d_neighbors, neighbors.data(), 
                        neighborCount * sizeof(int), hipMemcpyHostToDevice));
    
    // Launch kernel to find best parent
    int blocks = (neighborCount + BLOCK_SIZE - 1) / BLOCK_SIZE;
    findBestParentKernel<<<blocks, BLOCK_SIZE>>>(
        data.d_nodeX, data.d_nodeY, data.d_nodeCost,
        d_neighbors, neighborCount, x, y, d_costToNew, d_minCost, d_bestParent);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Get result
    int bestParent;
    CUDA_CHECK(hipMemcpy(&bestParent, d_bestParent, sizeof(int), hipMemcpyDeviceToHost));
    
    // Free memory
    CUDA_CHECK(hipFree(d_neighbors));
    CUDA_CHECK(hipFree(d_costToNew));
    CUDA_CHECK(hipFree(d_minCost));
    CUDA_CHECK(hipFree(d_bestParent));
    
    return bestParent;
}

// Function to check collision using CUDA
bool checkCollisionCuda(RRTInformedCudaData& data, float x1, float y1, float x2, float y2) {
    if (data.h_obstacleCount == 0) {
        return false; // No obstacles, no collision
    }
    
    // Allocate memory for collision result
    bool* d_collisionResult;
    CUDA_CHECK(hipMalloc(&d_collisionResult, sizeof(bool)));
    CUDA_CHECK(hipMemset(d_collisionResult, 0, sizeof(bool)));
    
    // Calculate number of blocks needed
    int blocks = (data.h_obstacleCount + BLOCK_SIZE - 1) / BLOCK_SIZE;
    
    // Launch kernel to check collisions
    checkCollisionKernel<<<blocks, BLOCK_SIZE>>>(
        x1, y1, x2, y2,
        data.d_obstacleX, data.d_obstacleY, data.d_obstacleWidth, data.d_obstacleHeight,
        data.h_obstacleCount, d_collisionResult);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Get the result
    bool collisionResult;
    CUDA_CHECK(hipMemcpy(&collisionResult, d_collisionResult, sizeof(bool), hipMemcpyDeviceToHost));
    
    // Free memory
    CUDA_CHECK(hipFree(d_collisionResult));
    
    return collisionResult;
}

// Function to rewire the tree using CUDA
void rewireTreeCuda(RRTInformedCudaData& data, int newNodeIdx, const std::vector<int>& neighbors) {
    if (neighbors.empty()) {
        return;
    }
    
    int neighborCount = neighbors.size();
    
    // Allocate memory for device arrays
    int* d_neighbors;
    int* d_rewireFlags;
    
    CUDA_CHECK(hipMalloc(&d_neighbors, neighborCount * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_rewireFlags, neighborCount * sizeof(int)));
    
    // Copy neighbors to device
    CUDA_CHECK(hipMemcpy(d_neighbors, neighbors.data(), 
                        neighborCount * sizeof(int), hipMemcpyHostToDevice));
    
    // Get new node data
    float newX, newY, newCost;
    CUDA_CHECK(hipMemcpy(&newX, &data.d_nodeX[newNodeIdx], sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(&newY, &data.d_nodeY[newNodeIdx], sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(&newCost, &data.d_nodeCost[newNodeIdx], sizeof(float), hipMemcpyDeviceToHost));
    
    // Launch kernel to check which neighbors should be rewired
    int blocks = (neighborCount + BLOCK_SIZE - 1) / BLOCK_SIZE;
    rewireNeighborsKernel<<<blocks, BLOCK_SIZE>>>(
        data.d_nodeX, data.d_nodeY, data.d_nodeCost, data.d_nodeParent,
        newNodeIdx, d_neighbors, neighborCount, newX, newY, newCost, d_rewireFlags);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Copy rewire flags back to host
    std::vector<int> h_rewireFlags(neighborCount);
    CUDA_CHECK(hipMemcpy(h_rewireFlags.data(), d_rewireFlags, 
                        neighborCount * sizeof(int), hipMemcpyDeviceToHost));
    
    // Apply rewiring on the host (for simplicity)
    for (int i = 0; i < neighborCount; i++) {
        if (h_rewireFlags[i] == 1) {
            int neighborIdx = neighbors[i];
            
            // Calculate new cost
            float dx = newX - data.h_nodes[neighborIdx].x;
            float dy = newY - data.h_nodes[neighborIdx].y;
            float edgeCost = sqrt(dx*dx + dy*dy);
            float newNeighborCost = newCost + edgeCost;
            
            // Update node cost and parent on host
            data.h_nodes[neighborIdx].cost = newNeighborCost;
            data.h_nodes[neighborIdx].parent = newNodeIdx;
            
            // Update cost and parent on device
            CUDA_CHECK(hipMemcpy(&data.d_nodeCost[neighborIdx], &newNeighborCost, 
                                sizeof(float), hipMemcpyHostToDevice));
            CUDA_CHECK(hipMemcpy(&data.d_nodeParent[neighborIdx], &newNodeIdx, 
                                sizeof(int), hipMemcpyHostToDevice));
            
            // TODO: Recursive rewiring of descendants (if needed)
            // This would require a separate function to propagate cost changes
        }
    }
    
    // Free memory
    CUDA_CHECK(hipFree(d_neighbors));
    CUDA_CHECK(hipFree(d_rewireFlags));
}

// Function to sample from the informed subset
void sampleInformedSubsetCuda(RRTInformedCudaData& data, float& x, float& y, 
                             float xMin, float xMax, float yMin, float yMax, float goalBias) {
    // Allocate memory for result
    float* d_x;
    float* d_y;
    CUDA_CHECK(hipMalloc(&d_x, sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_y, sizeof(float)));
    
    // Launch kernel to generate random node in the informed subset
    sampleInformedSubsetKernel<<<1, 1>>>(
        data.d_randStates,
        data.h_startX, data.h_startY, data.h_goalX, data.h_goalY,
        data.h_bestCost, goalBias,
        xMin, xMax, yMin, yMax,
        d_x, d_y);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Get results
    CUDA_CHECK(hipMemcpy(&x, d_x, sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(&y, d_y, sizeof(float), hipMemcpyDeviceToHost));
    
    // Free memory
    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));
}

// Function to add a new node to the tree
void addNodeCuda(RRTInformedCudaData& data, float x, float y, int parent, float time, float cost) {
    if (data.h_nodeCount >= data.d_nodeCapacity) {
        std::cerr << "Error: Node capacity exceeded" << std::endl;
        return;
    }
    
    // Add to host vector for easier path extraction later
    data.h_nodes.push_back(Node(x, y, parent, time, cost));
    
    // Add to device arrays
    CUDA_CHECK(hipMemcpy(&data.d_nodeX[data.h_nodeCount], &x, sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(&data.d_nodeY[data.h_nodeCount], &y, sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(&data.d_nodeParent[data.h_nodeCount], &parent, sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(&data.d_nodeTime[data.h_nodeCount], &time, sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(&data.d_nodeCost[data.h_nodeCount], &cost, sizeof(float), hipMemcpyHostToDevice));
    
    data.h_nodeCount++;
}

// Update best cost when a better solution is found
void updateBestCostCuda(RRTInformedCudaData& data, float newCost, int goalNodeIndex) {
    if (newCost < data.h_bestCost) {
        data.h_bestCost = newCost;
        data.h_goalNodeIndex = goalNodeIndex;
    }
}

// Steer function (implemented on host for simplicity)
// Steer function is defined in cudaRRTKernels.h

// Extract path from start to goal
std::vector<Node> extractPathCuda(const RRTInformedCudaData& data, int goalIndex) {
    std::vector<Node> path;
    
    if (goalIndex < 0 || goalIndex >= data.h_nodeCount) {
        return path; // Empty path if goal index is invalid
    }
    
    int currentIndex = goalIndex;
    
    while (currentIndex != -1) {
        path.push_back(data.h_nodes[currentIndex]);
        currentIndex = data.h_nodes[currentIndex].parent;
    }
    
    std::reverse(path.begin(), path.end());
    return path;
}

// Save tree data to file for visualization
void saveTreeToFileCuda(const RRTInformedCudaData& data, const std::string& filename) {
    std::ofstream file(filename);
    if (!file.is_open()) {
        std::cerr << "Failed to open file: " << filename << std::endl;
        return;
    }
    
    // Write header
    file << "node_id,x,y,parent_id,time,cost" << std::endl;
    
    // Write node data
    for (int i = 0; i < data.h_nodeCount; i++) {
        file << i << ","
             << data.h_nodes[i].x << ","
             << data.h_nodes[i].y << ","
             << data.h_nodes[i].parent << ","
             << data.h_nodes[i].time << ","
             << data.h_nodes[i].cost << std::endl;
    }
    
    file.close();
    std::cout << "Tree data saved to " << filename << std::endl;
}

// Main CUDA Informed RRT* algorithm
std::vector<Node> buildRRTInformedCuda(
    const Node& start,
    const Node& goal,
    const std::vector<Obstacle>& obstacles,
    double stepSize,
    double goalThreshold,
    int maxIterations,
    double rewireRadius,
    double xMin,
    double xMax,
    double yMin,
    double yMax,
    const std::string& treeFilename,
    bool enableVisualization,
    int numThreads,
    bool stopAtFirstSolution
) {
    // Start timing
    auto startTime = std::chrono::high_resolution_clock::now();
    
    // Initialize CUDA data
    RRTInformedCudaData data;
    initCudaRRTInformed(data, maxIterations + 2, obstacles.size(), numThreads); // +2 for start and goal
    
    // Store start and goal coordinates
    data.h_startX = start.x;
    data.h_startY = start.y;
    data.h_goalX = goal.x;
    data.h_goalY = goal.y;
    
    // Add start node with cost 0
    addNodeCuda(data, start.x, start.y, -1, 0.0, 0.0);
    
    // Copy obstacles to device
    data.h_obstacleCount = obstacles.size();
    if (obstacles.size() > 0) {
        std::vector<float> h_obstacleX(obstacles.size());
        std::vector<float> h_obstacleY(obstacles.size());
        std::vector<float> h_obstacleWidth(obstacles.size());
        std::vector<float> h_obstacleHeight(obstacles.size());
        
        for (size_t i = 0; i < obstacles.size(); i++) {
            h_obstacleX[i] = obstacles[i].x;
            h_obstacleY[i] = obstacles[i].y;
            h_obstacleWidth[i] = obstacles[i].width;
            h_obstacleHeight[i] = obstacles[i].height;
        }
        
        CUDA_CHECK(hipMemcpy(data.d_obstacleX, h_obstacleX.data(), obstacles.size() * sizeof(float), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(data.d_obstacleY, h_obstacleY.data(), obstacles.size() * sizeof(float), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(data.d_obstacleWidth, h_obstacleWidth.data(), obstacles.size() * sizeof(float), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(data.d_obstacleHeight, h_obstacleHeight.data(), obstacles.size() * sizeof(float), hipMemcpyHostToDevice));
    }
    
    // Main loop
    for (int i = 0; i < maxIterations; i++) {
        // Get current time for visualization
        auto currentTime = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed = currentTime - startTime;
        float timeSeconds = static_cast<float>(elapsed.count());
        
        // Sample from the informed subset (5% goal bias)
        float randomX, randomY;
        sampleInformedSubsetCuda(data, randomX, randomY, xMin, xMax, yMin, yMax, 0.05f);
        
        // Skip if inside obstacle
        bool insideObstacle = false;
        for (const auto& obstacle : obstacles) {
            if (randomX >= obstacle.x && randomX <= obstacle.x + obstacle.width &&
                randomY >= obstacle.y && randomY <= obstacle.y + obstacle.height) {
                insideObstacle = true;
                break;
            }
        }
        
        if (insideObstacle) {
            continue;
        }
        
        // Find nearest node
        int nearestIndex = findNearestCuda(data, randomX, randomY);
        
        // Skip if nearest node not found (should not happen)
        if (nearestIndex < 0) {
            continue;
        }
        
        // Steer towards random node
        float newX, newY;
        steerCuda(data.h_nodes[nearestIndex].x, data.h_nodes[nearestIndex].y, 
                 randomX, randomY, stepSize, newX, newY);
        
        // Check for collision
        if (checkCollisionCuda(data, data.h_nodes[nearestIndex].x, data.h_nodes[nearestIndex].y, 
                              newX, newY)) {
            continue; // Skip if collision
        }
        
        // Find nearby nodes for potential rewiring
        std::vector<int> nearNodes = findNodesInRadiusCuda(data, newX, newY, rewireRadius);
        
        // Choose best parent
        int bestParentIndex = chooseBestParentCuda(data, newX, newY, nearNodes);
        
        if (bestParentIndex < 0) {
            continue; // Skip if no valid parent found
        }
        
        // Calculate cost from start to this node
        float edgeCost = sqrtf(
            (newX - data.h_nodes[bestParentIndex].x) * (newX - data.h_nodes[bestParentIndex].x) +
            (newY - data.h_nodes[bestParentIndex].y) * (newY - data.h_nodes[bestParentIndex].y)
        );
        float nodeCost = data.h_nodes[bestParentIndex].cost + edgeCost;
        
        // Add new node
        addNodeCuda(data, newX, newY, bestParentIndex, timeSeconds, nodeCost);
        int newNodeIndex = data.h_nodeCount - 1;
        
        // Rewire the tree to optimize paths
        rewireTreeCuda(data, newNodeIndex, nearNodes);
        
        // Check if goal reached
        float dx = newX - goal.x;
        float dy = newY - goal.y;
        float distToGoal = sqrtf(dx*dx + dy*dy);
        
        if (distToGoal <= goalThreshold) {
            // Check if path to goal is collision-free
            if (!checkCollisionCuda(data, newX, newY, goal.x, goal.y)) {
                // Calculate total cost to goal
                float edgeCostToGoal = sqrtf(dx*dx + dy*dy);
                float totalCost = nodeCost + edgeCostToGoal;
                
                // Check if this is a better path
                if (totalCost < data.h_bestCost) {
                    // Add or update goal node with improved cost
                    if (data.h_goalNodeIndex >= 0) {
                        // Update existing goal node
                        data.h_nodes[data.h_goalNodeIndex].parent = newNodeIndex;
                        data.h_nodes[data.h_goalNodeIndex].cost = totalCost;
                        data.h_nodes[data.h_goalNodeIndex].time = timeSeconds;
                        
                        // Update on device
                        CUDA_CHECK(hipMemcpy(&data.d_nodeParent[data.h_goalNodeIndex], &newNodeIndex, sizeof(int), hipMemcpyHostToDevice));
                        CUDA_CHECK(hipMemcpy(&data.d_nodeCost[data.h_goalNodeIndex], &totalCost, sizeof(float), hipMemcpyHostToDevice));
                        CUDA_CHECK(hipMemcpy(&data.d_nodeTime[data.h_goalNodeIndex], &timeSeconds, sizeof(float), hipMemcpyHostToDevice));
                    } else {
                        // Add new goal node
                        addNodeCuda(data, goal.x, goal.y, newNodeIndex, timeSeconds, totalCost);
                        data.h_goalNodeIndex = data.h_nodeCount - 1;
                    }
                    
                    // Update best cost
                    updateBestCostCuda(data, totalCost, data.h_goalNodeIndex);
                    
                    std::cout << "Improved solution found with cost: " << totalCost << std::endl;
                    
                    // Stop at first solution if requested
                    if (stopAtFirstSolution) {
                        // Save the tree data if visualization is enabled
                        if (enableVisualization) {
                            saveTreeToFileCuda(data, treeFilename);
                        }
                        
                        std::cout << "Stopping at first solution as requested." << std::endl;
                        
                        // Extract and return path
                        std::vector<Node> path = extractPathCuda(data, data.h_goalNodeIndex);
                        
                        // Clean up CUDA resources
                        cleanupCudaRRTInformed(data);
                        
                        return path;
                    }
                }
            }
        }
        
        // Periodic visualization
        if (enableVisualization && i % 100 == 0) {
            saveTreeToFileCuda(data, treeFilename);
        }
    }
    
    // Save final tree data if visualization is enabled
    if (enableVisualization) {
        saveTreeToFileCuda(data, treeFilename);
    }
    
    // Extract path if goal was reached
    std::vector<Node> path;
    if (data.h_goalNodeIndex >= 0) {
        path = extractPathCuda(data, data.h_goalNodeIndex);
        
        // Print final solution information
        std::cout << "Final solution cost: " << data.h_bestCost << std::endl;
        std::cout << "Path length: " << path.size() << " nodes" << std::endl;
    } else {
        std::cout << "Goal not reached within max iterations." << std::endl;
    }
    
    // Clean up CUDA resources
    cleanupCudaRRTInformed(data);
    
    return path;
}