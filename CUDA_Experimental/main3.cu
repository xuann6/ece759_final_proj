
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <cmath>      // For sqrtf, floorf
#include <limits>     // For HUGE_VALF
#include <chrono>
#include <algorithm>  // For std::max, std::min, std::reverse
#include <hiprand/hiprand_kernel.h> // Include cuRAND for GPU random numbers
#include <math.h>     // For ceilf

// --- CUDA Error Checking Macro ---
#define CUDA_CHECK(err) { \
    hipError_t err_ = (err); \
    if (err_ != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err_)); \
        exit(EXIT_FAILURE); \
    } \
}

// --- Parameters ---
// World parameters
#define WORLD_WIDTH 100.0f
#define WORLD_HEIGHT 100.0f
#define STEP_SIZE 0.1f // Step size for tree growth AND connection attempts
#define START_X 10.0f
#define START_Y 10.0f
#define GOAL_X 90.0f
#define GOAL_Y 90.0f
// Goal threshold not directly used in Bi-RRT, connection is the goal

// RRT parameters
// Goal bias less relevant, trees grow towards each other
#define MAX_ITERATIONS 10000 // Max kernel launches (iterations)
#define MAX_NODES_PER_TREE 5000000 // Max nodes per tree
#define MAX_TOTAL_NODES (MAX_NODES_PER_TREE * 2)

// Obstacle parameters
#define OBSTACLE_WIDTH (WORLD_WIDTH / 10.0f)
#define OBSTACLE1_X (WORLD_WIDTH / 3.0f - OBSTACLE_WIDTH / 2.0f)
#define OBSTACLE1_Y 0.0f
#define OBSTACLE1_HEIGHT (0.6f * WORLD_HEIGHT)
#define OBSTACLE2_X (2.0f * WORLD_WIDTH / 3.0f - OBSTACLE_WIDTH / 2.0f)
#define OBSTACLE2_HEIGHT (0.6f * WORLD_HEIGHT)
#define OBSTACLE2_Y (WORLD_HEIGHT - OBSTACLE2_HEIGHT)

// CUDA execution parameters
#define THREADS_PER_BLOCK 128
// Use sufficient blocks for parallelism
#define NUM_BLOCKS 1

// --- Data Structures ---
// Node structure remains the same
typedef struct {
    float x;
    float y;
    int parent_idx; // Index of the parent node (-1 for root nodes)
} Node;

// Obstacle structure remains the same
typedef struct {
    float x_min;
    float y_min;
    float x_max;
    float y_max;
} Obstacle;

// --- Tree Indices ---
// Define indices for the two trees
#define START_TREE_IDX 0
#define GOAL_TREE_IDX 1


// --- Device Helper Functions ---

// Calculate squared Euclidean distance
__device__ inline float distance_sq(float x1, float y1, float x2, float y2) {
    float dx = x1 - x2;
    float dy = y1 - y2;
    return dx * dx + dy * dy;
}

// Clamp coordinates to world boundaries
__device__ inline float clamp(float val, float min_val, float max_val) {
    return fminf(max_val, fmaxf(min_val, val));
}

// Collision Check: Line segment vs AABB obstacles (Same as original)
__device__ bool is_collision(float x1, float y1, float x2, float y2,
                             const Obstacle* obstacles, int num_obstacles)
{
    // Endpoint check
     for (int i = 0; i < num_obstacles; ++i) {
        if (x2 >= obstacles[i].x_min && x2 <= obstacles[i].x_max &&
            y2 >= obstacles[i].y_min && y2 <= obstacles[i].y_max) {
            return true;
        }
    }
    // Midpoint check (Simplified)
    float mid_x = (x1 + x2) / 2.0f;
    float mid_y = (y1 + y2) / 2.0f;
     for (int i = 0; i < num_obstacles; ++i) {
         if (mid_x >= obstacles[i].x_min && mid_x <= obstacles[i].x_max &&
             mid_y >= obstacles[i].y_min && mid_y <= obstacles[i].y_max) {
             return true;
         }
     }
    return false;
}


// --- CUDA Kernels ---

// Kernel to initialize cuRAND states (Same as original)
__global__ void initialize_rng(hiprandState *states, unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}


// Main Bi-directional RRT Iteration Kernel
__global__ void birrt_iteration_kernel(
    Node* d_nodes,              // Combined array for both trees
    int* d_node_count_start,    // Node count for start tree
    int* d_node_count_goal,     // Node count for goal tree
    const Obstacle* d_obstacles,
    int num_obstacles,
    hiprandState *rng_states,
    int* d_connection_made,     // Flag: 0 = no connection, 1 = connection found
    int* d_connection_node_idx_start, // Index of connection node in start tree
    int* d_connection_node_idx_goal   // Index of connection node in goal tree
)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState local_rng_state = rng_states[tid];

    // --- 1. Check if connection already found ---
    if (*d_connection_made == 1) {
        rng_states[tid] = local_rng_state;
        return;
    }

    // --- 2. Determine which tree this thread works on ---
    // Simple alternation: even threads work on start tree, odd on goal tree
    int target_tree = tid % 2; // 0 for start tree, 1 for goal tree

    // Get pointers and limits for the target tree
    int* d_node_count_target; // Pointer to the count for the tree we are growing
    int* d_node_count_other;  // Pointer to the count for the other tree
    int node_offset_target;   // Starting index for nodes of the target tree
    int node_offset_other;    // Starting index for nodes of the other tree
    int max_nodes_target;     // Max index allowed for the target tree

    if (target_tree == START_TREE_IDX) { // Grow Start Tree
        d_node_count_target = d_node_count_start;
        d_node_count_other = d_node_count_goal;
        node_offset_target = 0;
        node_offset_other = MAX_NODES_PER_TREE; // Goal tree starts at midpoint
        max_nodes_target = MAX_NODES_PER_TREE;
    } else { // Grow Goal Tree
        d_node_count_target = d_node_count_goal;
        d_node_count_other = d_node_count_start;
        node_offset_target = MAX_NODES_PER_TREE; // Goal tree starts at midpoint
        node_offset_other = 0;
        max_nodes_target = MAX_TOTAL_NODES;
    }

    // Read current node counts (potentially stale)
    int current_node_count_target = *d_node_count_target;
    int current_node_count_other = *d_node_count_other;

    // Check if target tree is valid and not full
    if (current_node_count_target <= 0 || current_node_count_target >= MAX_NODES_PER_TREE) {
        rng_states[tid] = local_rng_state;
        return; // Target tree invalid or full
    }
     // Check if other tree has at least one node (needed for connection attempts)
    if (current_node_count_other <= 0) {
         rng_states[tid] = local_rng_state;
         return; // Other tree not yet initialized
    }


    // --- 3. Sample q_rand ---
    // Simple uniform sampling for Bi-RRT
    float q_rand_x = hiprand_uniform(&local_rng_state) * WORLD_WIDTH;
    float q_rand_y = hiprand_uniform(&local_rng_state) * WORLD_HEIGHT;


    // --- 4. Find Nearest Neighbor (q_near) in the TARGET tree ---
    // Uses the original linear scan approach
    int nearest_node_local_idx = -1; // Index relative to the start of the target tree's nodes
    float min_dist_sq = HUGE_VALF;

    // Linear scan through existing nodes in the target tree
    for (int i = 0; i < current_node_count_target; ++i) {
        int node_global_idx = node_offset_target + i; // Calculate global index
        float dist_sq = distance_sq(d_nodes[node_global_idx].x, d_nodes[node_global_idx].y, q_rand_x, q_rand_y);
        if (dist_sq < min_dist_sq) {
            min_dist_sq = dist_sq;
            nearest_node_local_idx = i; // Store local index
        }
    }

    // Should always find a nearest node if count > 0
    if (nearest_node_local_idx == -1) {
        rng_states[tid] = local_rng_state;
        return;
    }

    int nearest_node_global_idx = node_offset_target + nearest_node_local_idx;
    Node q_near = d_nodes[nearest_node_global_idx]; // Get the nearest node details


    // --- 5. Steer from q_near towards q_rand to get q_new ---
    float dir_x = q_rand_x - q_near.x;
    float dir_y = q_rand_y - q_near.y;
    float mag_sq = dir_x * dir_x + dir_y * dir_y; // Squared magnitude
    float q_new_x, q_new_y;

    if (mag_sq <= STEP_SIZE * STEP_SIZE || mag_sq == 0.0f) { // If q_rand is closer than step_size or at same point
        q_new_x = q_rand_x;
        q_new_y = q_rand_y;
    } else { // Steer by STEP_SIZE
        float mag = sqrtf(mag_sq);
        float scale = STEP_SIZE / mag;
        q_new_x = q_near.x + dir_x * scale;
        q_new_y = q_near.y + dir_y * scale;
    }

    // Clamp q_new to world boundaries
    q_new_x = clamp(q_new_x, 0.0f, WORLD_WIDTH);
    q_new_y = clamp(q_new_y, 0.0f, WORLD_HEIGHT);


    // --- 6. Collision Check for the new segment (q_near -> q_new) ---
    if (!is_collision(q_near.x, q_near.y, q_new_x, q_new_y, d_obstacles, num_obstacles)) {

        // --- 7. Add Node to TARGET Tree (If Collision Free) ---
        // Atomically get the local index for the new node within its tree
        int new_node_local_idx = atomicAdd(d_node_count_target, 1);

        // Check if the target tree is full BEFORE writing node data
        if (new_node_local_idx < MAX_NODES_PER_TREE) {
            int new_node_global_idx = node_offset_target + new_node_local_idx;

            // Write the new node data
            d_nodes[new_node_global_idx].x = q_new_x;
            d_nodes[new_node_global_idx].y = q_new_y;
            // Parent index is the GLOBAL index of the nearest node found earlier
            d_nodes[new_node_global_idx].parent_idx = nearest_node_global_idx;

            // --- 8. Attempt Connection to the OTHER tree ---
            // Find nearest node (q_connect) in the OTHER tree to the new node (q_new)
            int connect_node_other_local_idx = -1;
            float min_dist_sq_other = HUGE_VALF;

            // Linear scan through the OTHER tree
            // Use the potentially stale count read earlier for the loop bound
            for (int i = 0; i < current_node_count_other; ++i) {
                 int node_other_global_idx = node_offset_other + i;
                 float dist_sq = distance_sq(d_nodes[node_other_global_idx].x, d_nodes[node_other_global_idx].y, q_new_x, q_new_y);
                 if (dist_sq < min_dist_sq_other) {
                     min_dist_sq_other = dist_sq;
                     connect_node_other_local_idx = i; // Store local index relative to other tree
                 }
            }

            // If a nearest node was found in the other tree
            if (connect_node_other_local_idx != -1) {
                int connect_node_other_global_idx = node_offset_other + connect_node_other_local_idx;
                Node q_connect = d_nodes[connect_node_other_global_idx];

                // Check if q_new and q_connect are close enough (within STEP_SIZE)
                // AND the direct path between them is collision-free
                if (min_dist_sq_other <= STEP_SIZE * STEP_SIZE) { // Check distance first
                    if (!is_collision(q_new_x, q_new_y, q_connect.x, q_connect.y, d_obstacles, num_obstacles)) {
                        // --- Connection Successful! ---
                        // Atomically set the connection flag and store indices
                        if (atomicExch(d_connection_made, 1) == 0) {
                            // This thread is the first to make the connection
                            // Store the GLOBAL indices of the two connecting nodes
                            if (target_tree == START_TREE_IDX) {
                                atomicExch(d_connection_node_idx_start, new_node_global_idx);
                                atomicExch(d_connection_node_idx_goal, connect_node_other_global_idx);
                            } else { // target_tree == GOAL_TREE_IDX
                                atomicExch(d_connection_node_idx_start, connect_node_other_global_idx);
                                atomicExch(d_connection_node_idx_goal, new_node_global_idx);
                            }
                        }
                        // Even if connection made, let the node addition complete.
                        // The early exit at the start of the kernel will handle termination.
                    }
                }
            } // End if nearest node found in other tree

        } else { // Target tree was full when atomicAdd was executed
            // Revert the count increment
            atomicSub(d_node_count_target, 1);
        }
    } // End collision check block for adding q_new

    // --- Update RNG State ---
    rng_states[tid] = local_rng_state;
}


// --- Host Code ---
int main() {
    // --- Setup Obstacles ---
    std::vector<Obstacle> h_obstacles;
    h_obstacles.push_back({OBSTACLE1_X, OBSTACLE1_Y, OBSTACLE1_X + OBSTACLE_WIDTH, OBSTACLE1_Y + OBSTACLE1_HEIGHT});
    h_obstacles.push_back({OBSTACLE2_X, OBSTACLE2_Y, OBSTACLE2_X + OBSTACLE_WIDTH, OBSTACLE2_Y + OBSTACLE2_HEIGHT});
    int num_obstacles = h_obstacles.size();

    // --- Print Simulation Parameters ---
    printf("--- Bi-directional RRT Simulation Parameters ---\n");
    printf("World:       %.1f x %.1f\n", WORLD_WIDTH, WORLD_HEIGHT);
    printf("Start:       (%.1f, %.1f)\n", START_X, START_Y);
    printf("Goal:        (%.1f, %.1f)\n", GOAL_X, GOAL_Y);
    printf("Step Size:   %.2f\n", STEP_SIZE);
    printf("Obstacles:   %d\n", num_obstacles);
     for(int i=0; i<num_obstacles; ++i) {
        printf("  Obs %d:    (%.2f, %.2f) to (%.2f, %.2f)\n", i+1,
               h_obstacles[i].x_min, h_obstacles[i].y_min, h_obstacles[i].x_max, h_obstacles[i].y_max);
    }
    printf("Max Nodes/Tree: %d (Total: %d)\n", MAX_NODES_PER_TREE, MAX_TOTAL_NODES);
    printf("Max Iters:   %d\n", MAX_ITERATIONS);
    printf("--- CUDA Parameters ---\n");
    printf("Threads/Blk: %d\n", THREADS_PER_BLOCK);
    printf("Blocks:      %d\n", NUM_BLOCKS);
    printf("Total Thrds: %d\n", NUM_BLOCKS * THREADS_PER_BLOCK);
    printf("---------------------------------\n");


    // --- Host Memory Allocation ---
    Node* h_nodes = (Node*)malloc(MAX_TOTAL_NODES * sizeof(Node));
    if (!h_nodes) { fprintf(stderr, "Failed to allocate host memory for nodes.\n"); return 1; }

    // --- Device Memory Allocation ---
    Node* d_nodes;
    Obstacle* d_obstacles;
    hiprandState* d_rng_states;
    int* d_node_count_start; // Count for start tree
    int* d_node_count_goal;  // Count for goal tree
    int* d_connection_made;  // Connection flag
    int* d_connection_node_idx_start; // Connecting node index from start tree
    int* d_connection_node_idx_goal;  // Connecting node index from goal tree

    CUDA_CHECK(hipMalloc(&d_nodes, MAX_TOTAL_NODES * sizeof(Node)));
    CUDA_CHECK(hipMalloc(&d_obstacles, num_obstacles * sizeof(Obstacle)));
    CUDA_CHECK(hipMalloc(&d_rng_states, NUM_BLOCKS * THREADS_PER_BLOCK * sizeof(hiprandState)));
    CUDA_CHECK(hipMalloc(&d_node_count_start, sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_node_count_goal, sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_connection_made, sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_connection_node_idx_start, sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_connection_node_idx_goal, sizeof(int)));

    // --- Initialization ---
    printf("Initializing...\n");

    // Initialize start node (at index 0)
    h_nodes[0].x = START_X;
    h_nodes[0].y = START_Y;
    h_nodes[0].parent_idx = -1; // Root of start tree
    int h_node_count_start = 1;

    // Initialize goal node (at index MAX_NODES_PER_TREE)
    int goal_node_start_index = MAX_NODES_PER_TREE;
    h_nodes[goal_node_start_index].x = GOAL_X;
    h_nodes[goal_node_start_index].y = GOAL_Y;
    h_nodes[goal_node_start_index].parent_idx = -1; // Root of goal tree
    int h_node_count_goal = 1;

    // Initialize connection status
    int h_connection_made = 0;
    int h_connection_node_idx_start = -1;
    int h_connection_node_idx_goal = -1;

    // Copy initial data to device
    printf("  Copying initial data to device...\n");
    // Copy start node
    CUDA_CHECK(hipMemcpy(d_nodes, &h_nodes[0], sizeof(Node), hipMemcpyHostToDevice));
    // Copy goal node
    CUDA_CHECK(hipMemcpy(d_nodes + goal_node_start_index, &h_nodes[goal_node_start_index], sizeof(Node), hipMemcpyHostToDevice));
    // Copy obstacles
    CUDA_CHECK(hipMemcpy(d_obstacles, h_obstacles.data(), num_obstacles * sizeof(Obstacle), hipMemcpyHostToDevice));
    // Copy counts and flags
    CUDA_CHECK(hipMemcpy(d_node_count_start, &h_node_count_start, sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_node_count_goal, &h_node_count_goal, sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_connection_made, &h_connection_made, sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_connection_node_idx_start, &h_connection_node_idx_start, sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_connection_node_idx_goal, &h_connection_node_idx_goal, sizeof(int), hipMemcpyHostToDevice));

    // Initialize RNG states
    printf("  Initializing RNG states...\n");
    initialize_rng<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(d_rng_states, time(0));
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    printf("Initialization complete.\n");
    printf("Starting Bi-directional RRT...\n");
    auto start_time = std::chrono::high_resolution_clock::now();

    // --- Main Bi-RRT Loop ---
    int iteration;
    for (iteration = 0; iteration < MAX_ITERATIONS; ++iteration) {
        // Launch the Bi-RRT kernel
        birrt_iteration_kernel<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(
            d_nodes, d_node_count_start, d_node_count_goal,
            d_obstacles, num_obstacles, d_rng_states,
            d_connection_made, d_connection_node_idx_start, d_connection_node_idx_goal);

        // Periodically check if a connection has been made
        const int check_interval = 50; // Check more frequently than single RRT?
        if ((iteration + 1) % check_interval == 0 || iteration == MAX_ITERATIONS - 1) {
             CUDA_CHECK(hipDeviceSynchronize()); // Wait for kernel

             CUDA_CHECK(hipMemcpy(&h_connection_made, d_connection_made, sizeof(int), hipMemcpyDeviceToHost));
             if (h_connection_made == 1) {
                 CUDA_CHECK(hipMemcpy(&h_node_count_start, d_node_count_start, sizeof(int), hipMemcpyDeviceToHost));
                 CUDA_CHECK(hipMemcpy(&h_node_count_goal, d_node_count_goal, sizeof(int), hipMemcpyDeviceToHost));
                 printf("\nConnection established after %d iterations! Nodes: Start=%d, Goal=%d\n",
                        iteration + 1, h_node_count_start, h_node_count_goal);
                 break; // Exit loop
             }

             // Optional: Print progress
             const int print_interval = 500;
             if ((iteration + 1) % print_interval == 0) {
                CUDA_CHECK(hipMemcpy(&h_node_count_start, d_node_count_start, sizeof(int), hipMemcpyDeviceToHost));
                CUDA_CHECK(hipMemcpy(&h_node_count_goal, d_node_count_goal, sizeof(int), hipMemcpyDeviceToHost));
                printf("Iteration %d, Nodes: Start=%d, Goal=%d\n",
                       iteration + 1, h_node_count_start, h_node_count_goal);
             }

             // Check if either tree is full (read counts again if not read for print)
              if ((iteration + 1) % print_interval != 0) {
                   CUDA_CHECK(hipMemcpy(&h_node_count_start, d_node_count_start, sizeof(int), hipMemcpyDeviceToHost));
                   CUDA_CHECK(hipMemcpy(&h_node_count_goal, d_node_count_goal, sizeof(int), hipMemcpyDeviceToHost));
              }
             if (h_node_count_start >= MAX_NODES_PER_TREE || h_node_count_goal >= MAX_NODES_PER_TREE) {
                 printf("\nMaximum nodes per tree (%d) reached.\n", MAX_NODES_PER_TREE);
                 break; // Exit loop
             }
        }
    }

    CUDA_CHECK(hipDeviceSynchronize()); // Final sync

    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> duration_ms = end_time - start_time;
    printf("Bi-RRT computation finished in %.3f ms.\n", duration_ms.count());


    // --- Path Reconstruction (if connection made) ---
    CUDA_CHECK(hipMemcpy(&h_connection_made, d_connection_made, sizeof(int), hipMemcpyDeviceToHost));
    std::vector<int> path_indices; // Store final path indices

    if (h_connection_made) {
        // Get final counts and connection indices
        CUDA_CHECK(hipMemcpy(&h_node_count_start, d_node_count_start, sizeof(int), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(&h_node_count_goal, d_node_count_goal, sizeof(int), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(&h_connection_node_idx_start, d_connection_node_idx_start, sizeof(int), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(&h_connection_node_idx_goal, d_connection_node_idx_goal, sizeof(int), hipMemcpyDeviceToHost));

        printf("Reconstructing path from connection nodes: Start Tree Idx=%d, Goal Tree Idx=%d\n",
               h_connection_node_idx_start, h_connection_node_idx_goal);
        printf("Total Nodes: Start=%d, Goal=%d\n", h_node_count_start, h_node_count_goal);

        // Copy necessary nodes back from device
        // We need nodes from index 0 to h_node_count_start
        // and nodes from MAX_NODES_PER_TREE to MAX_NODES_PER_TREE + h_node_count_goal
        printf("  Copying nodes from device...\n");
        CUDA_CHECK(hipMemcpy(h_nodes, d_nodes, h_node_count_start * sizeof(Node), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(h_nodes + MAX_NODES_PER_TREE,         // Destination offset on host
                              d_nodes + MAX_NODES_PER_TREE,         // Source offset on device
                              h_node_count_goal * sizeof(Node),
                              hipMemcpyDeviceToHost));

        // Reconstruct path from start tree up to connection node
        std::vector<int> path_start_segment;
        int current_idx = h_connection_node_idx_start;
        int path_steps = 0;
        const int MAX_PATH_STEPS = h_node_count_start + h_node_count_goal + 2; // Safety break

        printf("  Tracing start tree segment...\n");
        while (current_idx != -1 && path_steps < MAX_PATH_STEPS) {
             if (current_idx < 0 || current_idx >= MAX_NODES_PER_TREE) { // Validate index range
                 fprintf(stderr, "Error: Invalid node index %d in start tree during path reconstruction.\n", current_idx);
                 path_start_segment.clear(); break;
             }
            path_start_segment.push_back(current_idx);
            current_idx = h_nodes[current_idx].parent_idx;
            path_steps++;
        }
        if (current_idx != -1 || path_steps >= MAX_PATH_STEPS) { // Check for failure
            fprintf(stderr, "Error: Start path reconstruction failed or exceeded max steps.\n");
            path_start_segment.clear();
        } else {
            std::reverse(path_start_segment.begin(), path_start_segment.end()); // Reverse to get start -> connection order
        }

        // Reconstruct path from goal tree up to connection node
        std::vector<int> path_goal_segment;
        current_idx = h_connection_node_idx_goal;
        path_steps = 0; // Reset step count

        printf("  Tracing goal tree segment...\n");
        while (current_idx != -1 && path_steps < MAX_PATH_STEPS) {
             // Validate index range for goal tree nodes (offset on host/device)
             if (current_idx < MAX_NODES_PER_TREE || current_idx >= MAX_TOTAL_NODES) {
                 fprintf(stderr, "Error: Invalid node index %d in goal tree during path reconstruction.\n", current_idx);
                 path_goal_segment.clear(); break;
             }
            path_goal_segment.push_back(current_idx);
            current_idx = h_nodes[current_idx].parent_idx; // Parent index is also global
            path_steps++;
        }
         if (current_idx != -1 || path_steps >= MAX_PATH_STEPS) { // Check for failure
            fprintf(stderr, "Error: Goal path reconstruction failed or exceeded max steps.\n");
            path_goal_segment.clear();
        }
        // NOTE: Goal segment is already in connection -> goal order, no need to reverse.

        // Combine the paths if both segments are valid
        if (!path_start_segment.empty() && !path_goal_segment.empty()) {
            path_indices = path_start_segment;
            path_indices.insert(path_indices.end(), path_goal_segment.begin(), path_goal_segment.end());
            printf("Path Found (%d steps).\n", (int)path_indices.size());
        } else {
            printf("Path reconstruction failed (one or both segments invalid).\n");
            h_connection_made = 0; // Mark as failed
        }

    } else {
        // Connection not made message
         CUDA_CHECK(hipMemcpy(&h_node_count_start, d_node_count_start, sizeof(int), hipMemcpyDeviceToHost));
         CUDA_CHECK(hipMemcpy(&h_node_count_goal, d_node_count_goal, sizeof(int), hipMemcpyDeviceToHost));
        if (iteration >= MAX_ITERATIONS) {
             printf("Connection not established within %d iterations. Nodes: Start=%d, Goal=%d\n",
                    MAX_ITERATIONS, h_node_count_start, h_node_count_goal);
        } else if (h_node_count_start >= MAX_NODES_PER_TREE || h_node_count_goal >= MAX_NODES_PER_TREE){
             printf("Connection not established. Tree reached maximum size. Nodes: Start=%d, Goal=%d\n",
                    h_node_count_start, h_node_count_goal);
        } else {
            printf("Connection not established. Status: iterations=%d, Nodes: Start=%d, Goal=%d\n",
                   iteration, h_node_count_start, h_node_count_goal);
        }
    }

    // --- Save results to CSV ---
    // Get final counts if not already retrieved
    if(h_node_count_start <= 1 || h_node_count_goal <= 1) {
         CUDA_CHECK(hipMemcpy(&h_node_count_start, d_node_count_start, sizeof(int), hipMemcpyDeviceToHost));
         CUDA_CHECK(hipMemcpy(&h_node_count_goal, d_node_count_goal, sizeof(int), hipMemcpyDeviceToHost));
    }
    // Copy nodes if connection wasn't made but we want to save the trees
    if (!h_connection_made) {
         printf("Copying nodes from device for saving trees...\n");
         if(h_node_count_start > 0)
            CUDA_CHECK(hipMemcpy(h_nodes, d_nodes, h_node_count_start * sizeof(Node), hipMemcpyDeviceToHost));
         if(h_node_count_goal > 0)
            CUDA_CHECK(hipMemcpy(h_nodes + MAX_NODES_PER_TREE, d_nodes + MAX_NODES_PER_TREE, h_node_count_goal * sizeof(Node), hipMemcpyDeviceToHost));
    }

    printf("Saving results to CSV files...\n");

    // Save node data
    FILE* nodes_file = fopen("birrt_nodes.csv", "w");
    if (nodes_file == NULL) {
        fprintf(stderr, "Error opening nodes output file 'birrt_nodes.csv'\n");
    } else {
        fprintf(nodes_file, "global_id,x,y,parent_global_id,tree_type,on_path\n"); // Added tree_type
        std::vector<bool> on_path(MAX_TOTAL_NODES, false); // Use global size
        if (h_connection_made && !path_indices.empty()) {
            for (int idx : path_indices) {
                if (idx >= 0 && idx < MAX_TOTAL_NODES) { // Check global index bounds
                    on_path[idx] = true;
                }
            }
        }

        // Write start tree nodes
        for (int i = 0; i < h_node_count_start; ++i) {
             int global_idx = i; // Start tree nodes are directly indexed
             fprintf(nodes_file, "%d,%.4f,%.4f,%d,%d,%d\n",
                     global_idx, h_nodes[global_idx].x, h_nodes[global_idx].y,
                     h_nodes[global_idx].parent_idx,
                     START_TREE_IDX, // Tree type 0
                     on_path[global_idx] ? 1 : 0);
        }
        // Write goal tree nodes
        for (int i = 0; i < h_node_count_goal; ++i) {
             int global_idx = MAX_NODES_PER_TREE + i; // Goal tree nodes are offset
             fprintf(nodes_file, "%d,%.4f,%.4f,%d,%d,%d\n",
                     global_idx, h_nodes[global_idx].x, h_nodes[global_idx].y,
                     h_nodes[global_idx].parent_idx,
                     GOAL_TREE_IDX, // Tree type 1
                     on_path[global_idx] ? 1 : 0);
        }
        fclose(nodes_file);
        printf("Saved %d start tree nodes and %d goal tree nodes to birrt_nodes.csv\n",
               h_node_count_start, h_node_count_goal);
    }

    // Save world data (Same format as before)
    FILE* world_file = fopen("birrt_world.csv", "w");
     if (world_file == NULL) {
        fprintf(stderr, "Error opening world output file 'birrt_world.csv'\n");
    } else {
        fprintf(world_file, "type,x,y,width,height_or_threshold\n");
        fprintf(world_file, "world,0,0,%.2f,%.2f\n", WORLD_WIDTH, WORLD_HEIGHT);
        fprintf(world_file, "start,%.2f,%.2f,0,0\n", START_X, START_Y);
        fprintf(world_file, "goal,%.2f,%.2f,0,0\n", GOAL_X, GOAL_Y); // Threshold not as relevant here
        for (size_t i = 0; i < h_obstacles.size(); i++) {
            fprintf(world_file, "obstacle,%.2f,%.2f,%.2f,%.2f\n",
                    h_obstacles[i].x_min, h_obstacles[i].y_min,
                    h_obstacles[i].x_max - h_obstacles[i].x_min,
                    h_obstacles[i].y_max - h_obstacles[i].y_min);
        }
        fclose(world_file);
        printf("Saved world data to birrt_world.csv\n");
    }


    // --- Cleanup ---
    printf("Cleaning up memory...\n");
    free(h_nodes);
    CUDA_CHECK(hipFree(d_nodes));
    CUDA_CHECK(hipFree(d_obstacles));
    CUDA_CHECK(hipFree(d_rng_states));
    CUDA_CHECK(hipFree(d_node_count_start));
    CUDA_CHECK(hipFree(d_node_count_goal));
    CUDA_CHECK(hipFree(d_connection_made));
    CUDA_CHECK(hipFree(d_connection_node_idx_start));
    CUDA_CHECK(hipFree(d_connection_node_idx_goal));
    // CUDA_CHECK(cudaDeviceReset()); // Optional

    printf("Done.\n");
    return 0;
}
